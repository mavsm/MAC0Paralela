#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>


//9 blocos, cada um d N/2 threads

__global__ 
void findMin(int N, int *S, int *min) {
	int tId = threadIdx.x; //thread ID
	int bId = blockIdx.x; //block ID

	int roundAnt = N;
	//A cada round toda thread tem um "companheiro" na outra metade de round. Desse modo todo elemento é checado
	for(unsigned int round=N/2; round>0; round/=2, roundAnt/=2) {
		if(tId < round) {
			if(S[tId*9 + bId] > S[(tId+round)*9+bId])
				S[tId*9+bId] = S[(tId+round)*9+bId];
		}

		if(tId == 0 && roundAnt%2 != 0){
			if(S[bId] > S[(roundAnt-1)*9+bId])
				S[bId] = S[(roundAnt-1)*9+bId];
		}
		__syncthreads();
	}
	if(tId == 0)
		min[bId] = S[bId];

}

int main(int argc, char **argv) {
	FILE *lista;
	int numMatrix, i;
	int *S; //N matrizes 3x3
	int *min;
	int threadsPerBlock, blockNum;
	char aux[3];

	//inicializa
	if(argc != 2) {
		printf("Número errado de argumentos!!\n");
		return 0;
	}
	lista = fopen(argv[1], "r");

	fscanf(lista, "%d", &numMatrix);

	hipMallocManaged(&S, 9*numMatrix*sizeof(int));
	hipMallocManaged(&min, 9*sizeof(int));

	for(i=0;i<numMatrix*9; i+=9) {
		fscanf(lista, "%s", aux);
		fscanf(lista, "%d %d %d", &S[i], &S[i+1], &S[i+2]);
		fscanf(lista, "%d %d %d", &S[i+3], &S[i+1+3], &S[i+2+3]);
		fscanf(lista, "%d %d %d", &S[i+6], &S[i+1+6], &S[i+2+6]);
	}

	//executa
	threadsPerBlock = numMatrix/2;
	blockNum = 9;

	//PARA RODAR NA REDE LINUX
	//cudaSetDevice(0);

	findMin<<< blockNum, threadsPerBlock >>>(numMatrix, S, min);
	hipDeviceSynchronize();

	printf("%d %d %d\n", min[0], min[1], min[2]);
	printf("%d %d %d\n", min[3], min[4], min[5]);
	printf("%d %d %d\n", min[6], min[7], min[8]);
		

	//free
	hipFree(S);
	hipFree(min);

	return 0;
}