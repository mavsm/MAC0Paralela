#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>


//9 blocos, cada um d N/2 threads

__global__ 
void findMin(int N, int *S, int *min) {
	int tId = threadIdx.x;
	int bId = blockIdx.x;

	//A cada round toda thread tem um "companheiro" na outra metade de round. Desse modo todo elemento é checado
	for(unsigned int round=N/2; round>0; round/=2) {
		if(tId < round) {
			if(S[tId*9 + bId] > S[(tId+round)*9+bId])
				S[tId*9+bId] = S[(tId+round)*9+bId];
		}

		if(tId == 0 && round%2 != 0){
			if(S[bId] > S[round*9+bId + 9])
				S[bId] = S[round*9+bId+9];
		}
		if(tId == 0)
			printf("Sou %d e olhei para %d\ntId = %d, bId = %d, round = %d \n", tId*9 + bId, (tId+round)*9+bId, tId, bId, round);

		__syncthreads();
	}
	if(tId == 0)
		min[bId] = S[bId];

}

int main(int argc, char **argv) {
	FILE *lista;
	int numMatrix, i;
	int *S; //N matrizes 3x3
	int *min;
	int threadsPerBlock, blockNum;
	char aux[3];

	//inicializa
	if(argc != 2) {
		printf("Número errado de argumentos!!\n");
		return 0;
	}
	lista = fopen(argv[1], "r");
	printf("Leu arquivo\n");

	fscanf(lista, "%d", &numMatrix);
	printf("%d\n", numMatrix);

	hipMallocManaged(&S, 9*numMatrix*sizeof(int));
	hipMallocManaged(&min, 9*sizeof(int));
	printf("Alocou memoria\n");

	for(i=0;i<numMatrix*9; i+=9) {
		fscanf(lista, "%s", aux);
		fscanf(lista, "%d %d %d", &S[i], &S[i+1], &S[i+2]);
		fscanf(lista, "%d %d %d", &S[i+3], &S[i+1+3], &S[i+2+3]);
		fscanf(lista, "%d %d %d", &S[i+6], &S[i+1+6], &S[i+2+6]);
	}
	printf("terminou leitura\n");

	//executa
	threadsPerBlock = numMatrix/2;
	blockNum = 9;

	//PARA RODAR NA REDE LINUX
	hipSetDevice(0);

	findMin<<< blockNum, threadsPerBlock >>>(numMatrix, S, min);
	hipDeviceSynchronize();

	printf("%d %d %d\n", min[0], min[1], min[2]);
	printf("%d %d %d\n", min[3], min[4], min[5]);
	printf("%d %d %d\n", min[6], min[7], min[8]);
		

	//free
	hipFree(S);
	hipFree(min);

	return 0;
}