#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>


//9 blocos, cada um d N/2 threads

__global__ 
void findMin(int N, int *S, int *min) {
	int tId = threadIdx.x;
	int bId = blockIdx.x;

	//A cada round toda thread tem um "companheiro" na outra metade de round. Desse modo todo elemento é checado
	for(unsigned int round=N/2; round>1; round/=2) {
		if(tId < round) {
			if(S[tId + bId] > S[tId+round*9+bId])
				S[tId+bId] = S[tId+round*9+bId];
		}

		if(tId == 0 && round%2 != 0){
			if(S[tId + bId] > S[tId+round*9+bId + 1])
				S[tId+bId] = S[tId+round*9+bId];
		}
		printf("Sou %d e olhei para %d\ntId = %d, bId = %d, round = %d \n", tId + bId, tId+round*9+bId, tId, bId, round);

		__syncthreads();
	}
	if(tId == 0)
		min[bId] = S[bId];

}

int main(int argc, char **argv) {
	FILE *lista;
	int numMatrix, i;
	int *S; //N matrizes 3x3
	int *min;
	int threadsPerBlock, blockNum;
	char aux[3];

	//inicializa
	if(argc != 2) {
		printf("Número errado de argumentos!!\n");
		return 0;
	}
	lista = fopen(argv[1], "r");
	printf("Leu arquivo\n");

	fscanf(lista, "%d", &numMatrix);
	printf("%d\n", numMatrix);

	hipMallocManaged(&S, 9*numMatrix*sizeof(int));
	hipMallocManaged(&min, 9*sizeof(int));
	printf("Alocou memoria\n");

	for(i=0;i<numMatrix*9; i+=9) {
		fscanf(lista, "%s", aux);
		fscanf(lista, "%d %d %d", &S[i], &S[i+1], &S[i+2]);
		fscanf(lista, "%d %d %d", &S[i+3], &S[i+1+3], &S[i+2+3]);
		fscanf(lista, "%d %d %d", &S[i+6], &S[i+1+6], &S[i+2+6]);
	}
	printf("terminou leitura\n");

	//executa
	threadsPerBlock = numMatrix/2;
	blockNum = 9;

	//PARA RODAR NA REDE LINUX
	hipSetDevice(0);

	findMin<<< blockNum, threadsPerBlock >>>(numMatrix, S, min);
	hipDeviceSynchronize();

	for(i=0; i<3; i+=1) {
		printf("%d %d %d\n", min[i], min[i+1], min[i+2]);
	}

	//free
	hipFree(S);
	hipFree(min);

	return 0;
}