#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <helper_functions.h>

int *S; //N matrizes 3x3
int *min;

//9 blocos, cada um d  threads, checa um dos 9 elementos [i][j] das matrizes

__global__ void findMin(int N) {
	int tId = threadIdx.x;
	int bId = blockId.x;

	for(unsigned int round=N/2; round>0; round/=2) {
		if(tId < round) {
			if(S[tId + bId] > S[tId+round*9+bId])
				S[tId+bId] = S[tId+round*9+bId];
		}

		if(tId == 0 && round%2 != 0){
			if(S[tId + bId] > S[tId+round*9+bId + 1])
				S[tId+bId] = S[tId+round*9+bId];
		}
		

		_syncthreads();
	}
	if(tId == 0)
		min[bId] = S[bId];

}

int main(int argc, char **argv) {
	FILE *lista;
	int numMatrix;
	int threadsPerBlock, blockNum;


	if(argc != 2) {
		printf("Número errado de argumentos!!\n");
		return 1;
	}

	lista = fopen(argv[1], "r");

	fscanf(lista, "%d", &numMatrix);

	hipMallocManaged(&S, 9*numMatrix*sizeof(int));
	hipMallocManaged(&min, 9*sizeof(int));

	threadsPerBlock = N/2;
	blockNum = 9;

	findMin<<< blockNum, threadsPerBlock >>>(numMatrix);
	hipDeviceSynchronize();


	hipFree(S);
	hipFree(min);

	return 0;
}