#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <limits.h>
#include <hip/hip_runtime.h>


//9 blocos, cada um d N/2 threads

__global__ 
void findMin(int N, int *S, int *min) {
	int tId = threadIdx.x;
	int bId = blockIdx.x;

	int roundAnt = N;

	if (tId > N/2)
		return;
	//A cada round toda thread tem um "companheiro" na outra metade de round. Desse modo todo elemento é checado
	for(unsigned int round=N/2; round>0; round/=2, roundAnt/=2) {
		if(tId < round) {
			if(S[tId*9 + bId] > S[(tId+round)*9+bId])
				S[tId*9+bId] = S[(tId+round)*9+bId];
		}

		if(tId == 0 && roundAnt%2 != 0){
			if(S[bId] > S[(roundAnt-1)*9+bId])
				S[bId] = S[(roundAnt-1)*9+bId];
		}
		__syncthreads();
	}
	if(tId == 0)
		if (S[bId] < min[bId])
		min[bId] = S[bId];

}

int main(int argc, char **argv) {
	FILE *lista;
	int numMatrixTotal, numMatrix, i, sizeOfS;
	int *S; //N matrizes 3x3
	int *min;
	int threadsPerBlock, blockNum;
	char aux[3];

	//inicializa
	if(argc != 2) {
		printf("Número errado de argumentos!!\n");
		return 0;
	}
	lista = fopen(argv[1], "r");

	fscanf(lista, "%d", &numMatrixTotal);
	numMatrix = numMatrixTotal;
	printf("%d\n", numMatrixTotal);

	hipMallocManaged(&min, 9*sizeof(int));
	for (i = 0; i < 9; i++)
		min[i] = INT_MAX;

	while(numMatrix > 0) {
		if (numMatrix > 2048) {
			sizeOfS = 2048;
			numMatrix -= 2048;
		}
		else {
			sizeOfS = numMatrix;
			numMatrix = 0;
		}

		hipMallocManaged(&S, 9*sizeOfS*sizeof(int));

		for(i=0;i<sizeOfS*9; i+=9) {
			fscanf(lista, "%s", aux);
			fscanf(lista, "%d %d %d", &S[i], &S[i+1], &S[i+2]);
			fscanf(lista, "%d %d %d", &S[i+3], &S[i+1+3], &S[i+2+3]);
			fscanf(lista, "%d %d %d", &S[i+6], &S[i+1+6], &S[i+2+6]);
		}

		//executa
		threadsPerBlock = sizeOfS/2;
		blockNum = 9;


		//PARA RODAR NA REDE LINUX
		//cudaSetDevice(0);
		printf("%d\n", threadsPerBlock);
		findMin<<< blockNum, threadsPerBlock >>>(numMatrixTotal, S, min);
		printf("%d", S[0]);
		hipDeviceSynchronize();
		hipFree(S);
		//break;
	}

	printf("%d %d %d\n", min[0], min[1], min[2]);
	printf("%d %d %d\n", min[3], min[4], min[5]);
	printf("%d %d %d\n", min[6], min[7], min[8]);
		

	//free
	hipFree(min);

	return 0;
}