#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <helper_functions.h>

int *S; //N matrizes 3x3
int *min;

//9 blocos, cada um d N/2 threads

__global__ void findMin(int N) {
	int tId = threadIdx.x;
	int bId = blockId.x;

	//A cada round toda thread tem um "companheiro" na outra metade de round. Desse modo todo elemento é checado
	for(unsigned int round=N/2; round>0; round/=2) {
		if(tId < round) {
			if(S[tId + bId] > S[tId+round*9+bId])
				S[tId+bId] = S[tId+round*9+bId];
		}

		if(tId == 0 && round%2 != 0){
			if(S[tId + bId] > S[tId+round*9+bId + 1])
				S[tId+bId] = S[tId+round*9+bId];
		}
		

		_syncthreads();
	}
	if(tId == 0)
		min[bId] = S[bId];

}

int main(int argc, char **argv) {
	FILE *lista;
	int numMatrix;
	int threadsPerBlock, blockNum;

	//inicializa
	if(argc != 2) {
		printf("Número errado de argumentos!!\n");
		return 1;
	}

	lista = fopen(argv[1], "r");

	fscanf(lista, "%d", &numMatrix);

	hipMallocManaged(&S, 9*numMatrix*sizeof(int));
	hipMallocManaged(&min, 9*sizeof(int));


	//executa
	threadsPerBlock = N/2;
	blockNum = 9;

	findMin<<< blockNum, threadsPerBlock >>>(numMatrix);
	hipDeviceSynchronize();

	//free
	hipFree(S);
	hipFree(min);

	return 0;
}