#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <helper_functions.h>

int *S; //N matrizes 3x3
int *min;

//9 blocos, cada um d N/2 threads

__global__ void findMin(int N) {
	int tId = threadIdx.x;
	int bId = blockId.x;

	//A cada round toda thread tem um "companheiro" na outra metade de round. Desse modo todo elemento é checado
	for(unsigned int round=N/2; round>0; round/=2) {
		if(tId < round) {
			if(S[tId + bId] > S[tId+round*9+bId])
				S[tId+bId] = S[tId+round*9+bId];
		}

		if(tId == 0 && round%2 != 0){
			if(S[tId + bId] > S[tId+round*9+bId + 1])
				S[tId+bId] = S[tId+round*9+bId];
		}
		

		_syncthreads();
	}
	if(tId == 0)
		min[bId] = S[bId];

}

int main(int argc, char **argv) {
	FILE *lista;
	int numMatrix;
	int threadsPerBlock, blockNum;

	//inicializa
	if(argc != 2) {
		printf("Número errado de argumentos!!\n");
		return 1;
	}

	lista = fopen(argv[1], "r");

	fscanf(lista, "%d", &numMatrix);

	hipMallocManaged(&S, 9*numMatrix*sizeof(int));
	hipMallocManaged(&min, 9*sizeof(int));


	for(i=0;i<numMatrix*9; i+=9) {
		fscanf(lista, "");
		fscanf(lista, "%d %d %d", &S[i] &S[i+1] &S[i+2]);
		fscanf(lista, "%d %d %d", &S[i+3] &S[i+1+3] &S[i+2+3]);
		fscanf(lista, "%d %d %d", &S[i+6] &S[i+1+6] &S[i+2+6]);
	}


	//executa
	threadsPerBlock = numMatrix/2;
	blockNum = 9;

	findMin<<< blockNum, threadsPerBlock >>>(numMatrix);
	hipDeviceSynchronize();

	for(i=0; i<3; i+=3)
		printf("%d %d %d\n", min[i] min[i+1] min[i+2])

	//free
	hipFree(S);
	hipFree(min);

	return 0;
}